/*
* Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>

// NOTE: These are cudf headers. Please be cautious.
// Using anything from these headers besides macros or typedefs
// will not work because this module is built before libcudf
// and therefore will not be able to link to any functions there.
// This module will be reworked appropriately in the future.
#include <cudf/utilities/error.hpp>

#include "nvstrings/NVStrings.h"
#include "nvstrings/NVText.h"

#include "../custring_view.cuh"
#include "../util.h"

struct stemmer_base_fn
{
    custring_view* d_vowels;
    Char y_char;

    stemmer_base_fn( custring_view* d_vowels, Char y_char )
    : d_vowels(d_vowels), y_char(y_char) {}

    __device__ bool is_consonant( custring_view* dstr, int index ) const
    {
        Char ch = dstr->at(index);
        if( d_vowels->find(ch) >= 0 )
            return false;
        if( (ch != y_char) || (index==0) )
            return true;
        ch = dstr->at(index-1);       // only if previous char
        return d_vowels->find(ch)>=0; // is not a consonant
    }
};

struct porter_stemmer_measure_fn : public stemmer_base_fn
{
    custring_view_array d_strings;
    unsigned int* d_results;

    porter_stemmer_measure_fn( custring_view* d_vowels, Char y_char,
                               custring_view_array d_strings, unsigned int* d_results )
    : stemmer_base_fn(d_vowels,y_char), d_strings(d_strings), d_results(d_results) {}

    __device__ void operator()(unsigned int idx)
    {
        custring_view* dstr = d_strings[idx];
        if( !dstr )
            return;
        unsigned int vcs = 0;
        bool vowel_run = !is_consonant(dstr,0);
        for( auto itr=dstr->begin(); itr!=dstr->end(); itr++ )
        {
            if( is_consonant(dstr,itr.position()) )
            {
                if( vowel_run )
                    vcs++;
                vowel_run = false;
            }
            else
                vowel_run = true;
        }
        d_results[idx] = vcs;
    }
};

unsigned int NVText::porter_stemmer_measure(NVStrings& strs, const char* vowels, const char* y_char, unsigned int* results, bool bdevmem )
{
    unsigned int count = strs.size();
    if( count==0 )
        return 0; // nothing to do
    auto execpol = rmm::exec_policy(0);
    // setup results vector
    unsigned int* d_results = results;
    if( !bdevmem )
        d_results = device_alloc<unsigned int>(count,0);
    if( vowels==nullptr )
        vowels = "aeiou";
    custring_view* d_vowels = custring_from_host(vowels);
    if( y_char==nullptr )
        y_char = "y";
    Char char_y;
    custring_view::char_to_Char(y_char,char_y);

    // get the string pointers
    rmm::device_vector<custring_view*> strings(count,nullptr);
    custring_view** d_strings = strings.data().get();
    strs.create_custring_index(d_strings);

    // do the measure
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        porter_stemmer_measure_fn{d_vowels,char_y,d_strings,d_results});

    // done
    if( !bdevmem )
    {
        CUDA_TRY( hipMemcpyAsync(results,d_results,count*sizeof(unsigned int),hipMemcpyDeviceToHost))
        RMM_FREE(d_results,0);
    }
    RMM_FREE(d_vowels,0);
    return 0;
}

//
unsigned int is_letter(NVStrings& strs, const char* vowels, const char* y_char,
                       NVText::letter_type ltype, int index, int* d_indices, bool* results, bool bdevmem )
{
    unsigned int count = strs.size();
    if( count==0 )
        return 0; // nothing to do
    auto execpol = rmm::exec_policy(0);
    // setup results vector
    bool* d_results = results;
    if( !bdevmem )
        d_results = device_alloc<bool>(count,0);
    if( vowels==nullptr )
        vowels = "aeiou";
    custring_view* d_vowels = custring_from_host(vowels);
    if( y_char==nullptr )
        y_char = "y";
    Char char_y;
    custring_view::char_to_Char(y_char,char_y);

    // get the string pointers
    rmm::device_vector<custring_view*> strings(count,nullptr);
    custring_view** d_strings = strings.data().get();
    strs.create_custring_index(d_strings);

    //
    stemmer_base_fn pfn{d_vowels,char_y};
    thrust::transform(execpol->on(0),
        thrust::make_counting_iterator<unsigned int>(0),
        thrust::make_counting_iterator<unsigned int>(count),
        d_results,
        [d_strings, pfn, ltype, index, d_indices] __device__ (unsigned int idx) {
            custring_view* d_str = d_strings[idx];
            if( !d_str )
                return false;
            int position = index;
            if( d_indices )
                position = d_indices[idx];
            int length = static_cast<int>(d_str->length());
            if( (position >= length) || (position < -length) )
                return false;
            position = (position + length) % length; // handles positive or negative index
            return pfn.is_consonant(d_str,position) ? ltype==NVText::consonant : ltype==NVText::vowel;
        });

    // done
    if( !bdevmem )
    {
        CUDA_TRY( hipMemcpyAsync(results,d_results,count*sizeof(bool),hipMemcpyDeviceToHost))
        RMM_FREE(d_results,0);
    }
    RMM_FREE(d_vowels,0);
    return 0;
}


// check individual characters are vowels or consonants
unsigned int NVText::is_letter(NVStrings& strs, const char* vowels, const char* y_char,
                               NVText::letter_type ltype, int position, bool* results, bool bdevmem )
{
    return ::is_letter(strs,vowels,y_char,ltype,position,nullptr,results,bdevmem);
}

//
unsigned int NVText::is_letter(NVStrings& strs, const char* vowels, const char* y_char,
                               NVText::letter_type ltype, int* d_indices, bool* results, bool bdevmem )
{
    return ::is_letter(strs,vowels,y_char,ltype,0,d_indices,results,bdevmem);
}
